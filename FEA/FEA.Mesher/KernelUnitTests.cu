#include "hip/hip_runtime.h"

// This file tests written cuda code for the Mesher Module

#include "Plane.cuh"
#include "Triangle.cuh"
#include "Vector.cuh"


extern "C" __global__ void TestCrossProduct(int Count,
Vector<float>* A,
Vector<float>* B,
Vector<float>* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i].Cross(B[i]);
	}

}

extern "C" __global__ void TestDotProduct(int Count,
Vector<float>* A,
Vector<float>* B,
float* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i].Dot(B[i]);
	}

}

extern "C" __global__ void TestAdd(int Count,
Vector<float>* A,
Vector<float>* B,
Vector<float>* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i] + B[i];
	}
}

extern "C" __global__ void TestSubtract(int Count,
Vector<float>* A,
Vector<float>* B,
Vector<float>* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i] - B[i];
	}
}

extern "C" __global__ void TestMultiply(int Count,
Vector<float>* A,
float* B,
Vector<float>* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i] * B[i];
	}
}

extern "C" __global__ void TestDivide(int Count,
Vector<float>* A,
float* B,
Vector<float>* C) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		C[i] = A[i] / B[i];
	}
}


extern "C" __global__ void TestLength(int Count,
Vector<float>* A,
float* B) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		B[i] = A[i].Length();
	}
}

extern "C" __global__ void TestNormalize(int Count,
Vector<float>* A,
Vector<float>* B) {

	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		B[i] = A[i];
		B[i].Normalize();
	}
}

extern "C" __global__ void TestTriangleArea(int Count,
Triangle<float>* A,
float* B) {
	
	int blockId = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.y * gridDim.z;
	int threadId = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.y * blockDim.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
	if (i < Count) {
		B[i] = A.Area();
	}
} 


